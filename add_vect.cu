#include "hip/hip_runtime.h"
//	Calling a simple CUDA kernel
 


#include "add_vect.h"


//Vector dot summation:  C = A + B (template)
template <typename ValueType>
__global__ void
vdAdd_kernel(const int N,
			 const ValueType * A, 
			 const ValueType * B,
			 ValueType * C)
{
	
	
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
	
}



//Add two double vectors on GPU
//Eventually change the type
template <>
void  add_vect(const int N, const double *A, const double *B, double *C)
{
		const int BLOCK_SIZE = 512;
		const int NUM_BLOCKS =  N/BLOCK_SIZE + (N%BLOCK_SIZE == 0 ? 0 : 1);
		
		vdAdd_kernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(N,A,B,C);
		
		
		return;
}
